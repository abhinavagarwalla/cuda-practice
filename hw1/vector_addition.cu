
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>

// error checking macro
#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)

__global__ void add(float* a, float* b, float* c, int size){
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if(idx < size) c[idx] = a[idx] + b[idx];
}

void print_array(float *c, int size){
    std::cout << "Result: \n";
    for(int i=0;i<size;++i) std::cout <<c[i] << ",";
    std::cout << std::endl;
    return;
}

#define VSIZE 1024
#define THREADS 512
int main(){
    float *a, *b, *c;
    a = new float[VSIZE];
    b = new float[VSIZE];
    c = new float[VSIZE];
    float *da, *db, *dc;

    for(int i=0;i<VSIZE;++i){
        a[i] = 1.0f * rand() / VSIZE;
        b[i] = 1.0f * rand() / VSIZE;
        c[i] = 0.0f;
    }
    //print_array(a, VSIZE);
    //print_array(b, VSIZE);

    hipMalloc(&da, VSIZE*sizeof(float));
    hipMalloc(&db, VSIZE*sizeof(float));
    hipMalloc(&dc, VSIZE*sizeof(float));

    cudaCheckErrors("hipMalloc error check.");

    hipMemcpy(da, a, VSIZE*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(db, b, VSIZE*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dc, c, VSIZE*sizeof(float), hipMemcpyHostToDevice);

    cudaCheckErrors("hipMemcpy error check.");

    add<<<(VSIZE + THREADS-1)/THREADS, THREADS>>>(da, db, dc, VSIZE);
    cudaCheckErrors("kernel execution failure");
    
    hipMemcpy(c, dc, VSIZE*sizeof(float), hipMemcpyDeviceToHost);
    hipFree(da);
    hipFree(db);
    hipFree(dc);

    
    print_array(c, VSIZE);

}
